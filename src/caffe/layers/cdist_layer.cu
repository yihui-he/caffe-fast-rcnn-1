#include "hip/hip_runtime.h"
#include <vector>

#include "caffe/layers/cdist_layer.hpp"
#include "caffe/util/math_functions.hpp"

namespace caffe {

  template <typename Dtype>
  __global__ void cdist_fwd_kernel(const int count, const int channels, const int height, const int width,
              const int out_channels, const int cos,
              const Dtype *input_data, const Dtype *weight_data, Dtype *output_data) { //, const Dtype *bias_data
    CUDA_KERNEL_LOOP(index, count) {
        output_data+=index;
        const int ow = index % width;
        index /= width;
        const int oh = index % height;
        index /= height;
        const int oc = index % out_channels;
        const int n = index / out_channels;
  
        const int iw = ow;
        const int ih = oh;
  
        input_data += ((n * channels ) * height + ih) * width + iw;
        weight_data += oc * channels;
  
        Dtype v = 0;
        if (cos) {
          for (int i = 0; i < channels; i++){
            v += (input_data [i*height*width] * weight_data[i]);
          }
          v = 1 - v;
        } else {
            for (int i = 0; i < channels; i++){
              v += (input_data [i*height*width] - weight_data[i]) * (input_data[i*height*width] - weight_data[i]);
            }
        }
  
        *output_data = v;
    }
  }


template <typename Dtype>
void cdistLayer<Dtype>::Forward_gpu(const vector<Blob<Dtype>*>& bottom,
    const vector<Blob<Dtype>*>& top) {
  const Dtype* bottom_data = bottom[0]->gpu_data();
  const Dtype* weight = bottom[1]->gpu_data();
  Dtype* top_data = top[0]->mutable_gpu_data();
  const int dim = top[0]->count();
  const int c = bottom[0]->shape(1);
  const int h = bottom[0]->shape(2); 
  const int w = bottom[0]->shape(3);
  const int oc  =bottom[1]->shape(0);
  CHECK_EQ(c, bottom[1]->shape(1));
  cdist_fwd_kernel<Dtype>  // NOLINT_NEXT_LINE(whitespace/operators)
        <<<CAFFE_GET_BLOCKS(dim), CAFFE_CUDA_NUM_THREADS>>>(
          dim, c, h,w,oc, cos_, bottom_data, weight, top_data);
}

template <typename Dtype>
void cdistLayer<Dtype>::Backward_gpu(const vector<Blob<Dtype>*>& top,
    const vector<bool>& propagate_down, const vector<Blob<Dtype>*>& bottom) {
  NOT_IMPLEMENTED;
}

INSTANTIATE_LAYER_GPU_FUNCS(cdistLayer);

}  // namespace caffe
